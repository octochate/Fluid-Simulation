#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>


#define IX(i,j) ((i)+(N+2)*(j))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}

__global__ void add_source(float* x, float* s, float dt, int size)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    if (idx < size) {
        x[idx] += dt * s[idx];
    }
}

__device__ void set_bnd(int N, int b, float* x, int index)
{
    if (index > N) {
        return;
    }
    int i = index;

    x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
    x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
    x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
    x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
    
    if (index == 0) {
        x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
        x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
        x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
        x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
    }
}
__global__ void lin_solve(int N, int b, float* x, float* x0, float a, float c)
{
    int i, j, k;
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx > (N + 2) * (N + 2)) {
        return;
    }

    j = idx / N;
    i = idx % N;

    for (k = 0; k < 20; k++)
    {
        x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
        __syncthreads();
         set_bnd(N, b, x, idx);
        __syncthreads();
    }
}


__global__ void advect(int N, int b, float* d, float* d0, float* u, float* v, float dt)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    if (idx > (N + 2) * (N + 2)) {
        return;
    }

    int j = idx / N;
    int i = idx % N;

    int i0, j0, i1, j1;
    float x, y, s0, t0, s1, t1, dt0;

    dt0 = dt * N;
        x = i - dt0 * u[IX(i, j)]; y = j - dt0 * v[IX(i, j)];
    if (x < 0.5f) x = 0.5f; if (x > N + 0.5f) x = N + 0.5f; i0 = (int)x; i1 = i0 + 1;
    if (y < 0.5f) y = 0.5f; if (y > N + 0.5f) y = N + 0.5f; j0 = (int)y; j1 = j0 + 1;
    s1 = x - i0; s0 = 1 - s1; t1 = y - j0; t0 = 1 - t1;
    d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
        s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);

    __syncthreads();
    set_bnd(N, b, d, idx);
}


__global__ void project1(int N, float* u, float* v, float* p, float* div)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    if (idx > (N + 2) * (N + 2)) {
        return;
    }

    int j = idx / N;
    int i = idx % N;

        div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / N;
        p[IX(i, j)] = 0;

        __syncthreads();
        set_bnd(N, 0, div, idx); 
        __syncthreads();
        set_bnd(N, 0, p, idx);
}

__global__ void project3(int N, float* u, float* v, float* p, float* div)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    if (idx > (N + 2) * (N + 2)) {
        return;
    }

    int j = idx / N;
    int i = idx % N;

        u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);

    __syncthreads();
    set_bnd(N, 1, u, idx); 
    __syncthreads();
    set_bnd(N, 2, v, idx);
}

void get_from_UI(float* d, float* u, float* v, int force, int source, int N)
{
    int i, j, size = (N + 2) * (N + 2);

    for (i = 0; i < size; i++)
    {
        u[i] = v[i] = d[i] = 0.0f;
    }

    i = N / 2;
    j = N / 2;

    if (i < 1 || i > N || j < 1 || j > N)
        return;

    u[IX(i, j)] = force * (1.0);
    v[IX(i, j)] = force * (1.0);
    d[IX(i, j)] = source * 1.0;

    return;
}

static void printFrameMatrices(float* dens, float* u, float* v, int N)
{
        printf("Density Matrix:\t\t\t\t\tVelocity U Matrix:\t\t\t\t\tVelocity V Matrix:\n");
        for (int j = 0; j < 1; j++)
        {
            printf("%f, ", dens[j * N]);
            printf("%f, ", dens[1 + j * N]);
            printf("%f, ", dens[2 + j * N]);
            printf("%f", dens[3 + j * N]);
            printf("\t\t");
            printf("%f, ", u[j * N]);
            printf("%f, ", u[1 + j * N]);
            printf("%f, ", u[2 + j * N]);
            printf("%f", u[3 + j * N]);
            if (j == 1)
            {
                printf("\t");
            }
            else
            {
                printf("\t\t");
            }
            if (j != 1)
            {
                printf("%f, ", v[j * N]);
                printf("%f, ", v[1 + j * N]);
                printf("%f, ", v[2 + j * N]);
                printf("%f", v[3 + j * N]);
            }
            else
            {
                printf("\t%f, ", v[j * N]);
                printf("%f, ", v[1 + j * N]);
                printf("%f, ", v[2 + j * N]);
                printf("%f", v[3 + j * N]);
            }
            printf("\n");
        }
        printf("\n");
}

int main(int argc, char* argv[]) {

    int N = 4;
    float dt = 0.1f, diff = 0.0f, visc = 0.0f;
    float force = 0.5f, source = 100.0f;

    // Allocate space for host
    int size = (N + 2) * (N + 2);

    // Device copies of data
    float* u_cuda, * v_cuda, * u_prev_cuda, * v_prev_cuda;
    float* dens_cuda, * dens_prev_cuda;

    // Allocate space for device copies
    hipMallocManaged((void**)&u_cuda, sizeof(float));
    hipMallocManaged((void**)&v_cuda, sizeof(float));
    hipMallocManaged((void**)&u_prev_cuda, sizeof(float));
    hipMallocManaged((void**)&v_prev_cuda, sizeof(float));
    hipMallocManaged((void**)&dens_cuda, sizeof(float));
    hipMallocManaged((void**)&dens_prev_cuda, sizeof(float));

    // have copy of host data
    get_from_UI(dens_cuda, u_cuda, v_cuda, force, source, N);


    // Parallelize computation :: TODO update values to match
    int num_threads = (N + 2);
    int num_blocks = (N + 2);

    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    // Velocity timestep parallelization
    add_source <<< num_blocks, num_threads, 0, stream1>>> (u_cuda, u_prev_cuda, dt, size);
    add_source <<< num_blocks, num_threads, 0, stream2>>> (v_cuda, v_prev_cuda, dt, size);
    add_source <<< num_blocks, num_threads, 0, stream3>>> (dens_cuda, dens_prev_cuda, dt, size);
    hipDeviceSynchronize();

    SWAP(u_prev_cuda, u_cuda);
    SWAP(v_prev_cuda, v_cuda);
    SWAP(dens_prev_cuda, dens_cuda);

    // diffuse
    float a = dt * visc * N * N;
    lin_solve << < num_blocks, num_threads >> > (N, 1, u_prev_cuda, u_cuda, a, 1 + 4 * a); // diffuse u_cuda
    hipDeviceSynchronize();

    lin_solve << < num_blocks, num_threads >> > (N, 2, v_prev_cuda, v_cuda, a, 1 + 4 * a); // diffuse v_cuda
    a = dt * diff * N * N;
    lin_solve << < num_blocks, num_threads >> > (N, 0, dens_prev_cuda, dens_cuda, a, 1 + 4 * a); // diffuse dens_cuda
    hipDeviceSynchronize();

    project1 << < num_blocks, num_threads >> > (N, u_cuda, v_cuda, u_prev_cuda, v_prev_cuda);
    hipDeviceSynchronize();

    lin_solve << < num_blocks, num_threads >> > (N, 0, u_prev_cuda, v_prev_cuda, 1, 4);
    hipDeviceSynchronize();

    project3 << < num_blocks, num_threads >> > (N, u_cuda, v_cuda, u_prev_cuda, v_prev_cuda);
    hipDeviceSynchronize();

    SWAP(dens_prev_cuda, dens_cuda);
    SWAP(u_prev_cuda, u_cuda);
    SWAP(v_prev_cuda, v_cuda);


    advect << < num_blocks, num_threads >> > (N, 1, u_cuda, u_prev_cuda, u_prev_cuda, v_prev_cuda, dt);
    advect << < num_blocks, num_threads >> > (N, 2, v_cuda, v_prev_cuda, u_prev_cuda, v_prev_cuda, dt);
    hipDeviceSynchronize();

    project1 << < num_blocks, num_threads >> > (N, u_cuda, v_cuda, u_prev_cuda, v_prev_cuda);
    hipDeviceSynchronize();
    lin_solve << < num_blocks, num_threads >> > (N, 0, u_prev_cuda, v_prev_cuda, 1, 4);
    hipDeviceSynchronize();
    project3 << < num_blocks, num_threads >> > (N, u_cuda, v_cuda, u_prev_cuda, v_prev_cuda);
    hipDeviceSynchronize();

    // Density timestep parallelization
    advect << < num_blocks, num_threads >> > (N, 0, dens_cuda, dens_prev_cuda, u_cuda, v_cuda, dt);
    hipDeviceSynchronize();

    // Copy result back to host
    printf("%f", dens_cuda[0]);

    // Free Device space
    hipFree(u_cuda);
    hipFree(v_cuda);
    hipFree(u_prev_cuda);
    hipFree(dens_cuda);
    hipFree(dens_prev_cuda);
    hipFree(v_prev_cuda);

    return 0;
}
