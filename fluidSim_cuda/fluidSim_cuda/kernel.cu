#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <time.h>

#define IX(i,j) ((i)+(N+2)*(j))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}

__global__ void add_source(float* x, float* s, float dt, int size)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    if (idx < size) {
        x[idx] += dt * s[idx];
    }
}

__device__ void set_bnd(int N, int b, float* x, int index, int elementsPerThread)
{
    int i = index + 1;
    if (i > N + 1) {
        return;
    }

    int size = (N + 2) * (N + 2);

    while (i < (index + elementsPerThread) && i <= N + 1) {
        x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
        x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
        i++;
    }

    __syncthreads();

    if (index == 0) {
        x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
        x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
        x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
        x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
    }
}

__global__ void lin_solve(int N, int b, float* x, float* x0, float a, float c, int elementPerThread)
{
    int i, j, k, idxNew;
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

    if (idx >= N * N) {
        return;
    }

    for (k = 0; k < 20; k++)
    {
        idxNew = idx;
        while (idxNew < idx + elementPerThread && idxNew < N * N) {
            j = idxNew / N + 1;
            i = idxNew % N + 1;
            x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
            idxNew++;
        }
        __syncthreads();
        set_bnd(N, b, x, idx, elementPerThread);
    }
}

__global__ void advect(int N, int b, float* d, float* d0, float* u, float* v, float dt, int elementPerThread)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

    if (idx >= N * N) {
        return;
    }

    int idxNew = idx;
    while (idxNew < idx + elementPerThread && idxNew < N * N) {
        int j = idxNew / N + 1;
        int i = idxNew % N + 1;

        int i0, j0, i1, j1;
        float x, y, s0, t0, s1, t1, dt0;

        dt0 = dt * N;
        x = i - dt0 * u[IX(i, j)]; y = j - dt0 * v[IX(i, j)];
        if (x < 0.5f) x = 0.5f; if (x > N + 0.5f) x = N + 0.5f; i0 = (int)x; i1 = i0 + 1;
        if (y < 0.5f) y = 0.5f; if (y > N + 0.5f) y = N + 0.5f; j0 = (int)y; j1 = j0 + 1;
        s1 = x - i0; s0 = 1 - s1; t1 = y - j0; t0 = 1 - t1;

        d[IX(i, j)] =  s0* (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
            s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);

        idxNew++;
    }
    
    __syncthreads();
    set_bnd(N, b, d, idx, elementPerThread);
}

__global__ void project1(int N, float* u, float* v, float* p, float* div, int elementPerThread)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

    if (idx >= N * N) {
        return;
    }

    int idxNew = idx;
    while (idxNew < idx + elementPerThread && idxNew < N * N) {
        int j = idxNew / N + 1;
        int i = idxNew % N + 1;

        div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / N;
        p[IX(i, j)] = 0;
        idxNew++;
    }

    __syncthreads();
    set_bnd(N, 0, div, idx, elementPerThread);
    set_bnd(N, 0, p, idx, elementPerThread);
}

__global__ void project3(int N, float* u, float* v, float* p, int elementPerThread)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

    if (idx >= N * N) {
        return;
    }

    int idxNew = idx;
    while (idxNew < idx + elementPerThread && idxNew < N * N) {
        int j = idxNew / N + 1;
        int i = idxNew % N + 1;

        u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
        v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);

        idxNew++;
    }
    __syncthreads();
    set_bnd(N, 1, u, idx, elementPerThread);
    set_bnd(N, 2, v, idx, elementPerThread);
}

void get_from_UI(float* d, float* u, float* v, int force, int source, int N)
{
    int i, j, size = (N + 2) * (N + 2);

    for (i = 0; i < size; i++)
    {
        u[i] = v[i] = d[i] = 0.0f;
    }

    i = (N+2) / 2;
    j = (N+2) / 2;

    if (i < 1 || i > N || j < 1 || j > N)
        return;

    u[IX(i, j)] = force * (1.0);
    v[IX(i, j)] = force * (1.0);
    d[IX(i, j)] = source * 1.0;

    return;
}

static void printFrameMatrices(float* dens, float* u, float* v, int N)
{
    printf("Density Matrix:\t\t\t\t\t\t\tVelocity U Matrix:\t\t\t\t\t\tVelocity V Matrix:\n");
    for (int i = 0; i <= N + 1; i++) {
        for (int j = 0; j <= N + 1; j++)
        {
            printf("%f, ", dens[IX(i, j)]);
        }
        printf("\t");
        for (int j = 0; j <= N + 1; j++)
        {
            printf("%f, ", u[IX(i, j)]);
        }
        printf("\t");
        for (int j = 0; j <= N + 1; j++)
        {
            printf("%f, ", v[IX(i, j)]);
        }
        printf("\n");
    }
}

int main(int argc, char* argv[]) {

    // declare variables
    float dt = 0.1f, diff = 0.1f, visc = 0.1f;
    float force = 100.0f, source = 100.0f;
    int N = 5, num_threads = 2, num_iterations = 10, display_output = 0, use_streams = 0;

    // load parameters from command line
    if (argc != 6) {
        fprintf(stderr, "usage : %s <grid size> <number of threads> <number of iterations> <display output> <use CUDA streams>\n", argv[0]);
        fprintf(stderr, "where:\n"); \
        fprintf(stderr, "\t grid_size                : grid resolution\n");
        fprintf(stderr, "\t number_of_threads        : number of threads running per kernel (1 block/kernel enforced)\n");
        fprintf(stderr, "\t number_of_iterations     : number of iterations to run the simulation\n");
        fprintf(stderr, "\t display_output           : Whether to display the opdated grids after every iteration\n");
        fprintf(stderr, "\t use_CUDA_streams         : Whether to use CUDA streams or have all kernels run sequentially\n");
        exit(1);
    }


    N = atoi(argv[1]);
    num_threads = atoi(argv[2]);
    num_iterations = atoi(argv[3]);
    display_output = atoi(argv[4]);
    use_streams = atoi(argv[5]);

    // validate parameters
    if (N <= 0) {
        fprintf(stderr, "grid_size must be an integer larger than 0\n");
        exit(1);
    }
    if (num_threads <= 0 || num_threads > 1024) {
        fprintf(stderr, "number_of_threads must be an integer in the range [1, 1024]\n");
        exit(1);
    }
    if (num_iterations <= 0) {
        fprintf(stderr, "number_of_iterations must be an integer larger than 0\n");
        exit(1);
    }
    if (display_output != 0 && display_output != 1) {
        fprintf(stderr, "display_output must be 1 if TRUE and 0 if FALSE\n");
        exit(1);
    }
    if (use_streams != 0 && use_streams != 1) {
        fprintf(stderr, "use_streams must be 1 if TRUE and 0 if FALSE\n");
        exit(1);
    }

    
    int size = (N + 2) * (N + 2);

    // unified memory: pointers to data
    float* u_cuda, * v_cuda, * u_prev_cuda, * v_prev_cuda;
    float* p_cuda, * div_cuda;
    float* dens_cuda, * dens_prev_cuda;

    // Allocate space for arrays
    hipMallocManaged(&u_cuda, sizeof(float) * size);
    hipMallocManaged(&v_cuda, sizeof(float) * size);
    hipMallocManaged(&p_cuda, sizeof(float) * size);
    hipMallocManaged(&div_cuda, sizeof(float) * size);
    hipMallocManaged(&u_prev_cuda, sizeof(float) * size);
    hipMallocManaged(&v_prev_cuda, sizeof(float) * size);
    hipMallocManaged(&dens_cuda, sizeof(float) * size);
    hipMallocManaged(&dens_prev_cuda, sizeof(float) * size);

    // Initialize all arrays to 0
    hipMemset(u_cuda, 0, sizeof(float) * size);
    hipMemset(v_cuda, 0, sizeof(float) * size);
    hipMemset(p_cuda, 0, sizeof(float) * size);
    hipMemset(div_cuda, 0, sizeof(float) * size);
    hipMemset(u_prev_cuda, 0, sizeof(float) * size);
    hipMemset(v_prev_cuda, 0, sizeof(float) * size);
    hipMemset(dens_cuda, 0, sizeof(float) * size);
    hipMemset(dens_prev_cuda, 0, sizeof(float) * size);


    // Parallelization scheme for add_source
    int num_threads_source = (N + 2);
    int num_blocks_source = (N + 2);
    
    // Parallelization scheme for only using one block
    int elementsPerThread = (N * N + 1) / num_threads;

    // Instantiate cuda streams
    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    if (use_streams) {
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);
    }
    else {
        stream2 = stream1;
        stream3 = stream1;
    }
    


    // update loop
    for (int i = 0; i < num_iterations; i++) {

        // get input from 'GUI'
        get_from_UI(dens_prev_cuda, u_prev_cuda, v_prev_cuda, force, source, N);


        //// Add GUI inputs to grids
        add_source <<< num_blocks_source, num_threads_source >>> (u_cuda, u_prev_cuda, dt, size);
        add_source <<< num_blocks_source, num_threads_source >>> (v_cuda, v_prev_cuda, dt, size);
        add_source <<< num_blocks_source, num_threads_source >>> (dens_cuda, dens_prev_cuda, dt, size);
        hipDeviceSynchronize();

        SWAP(u_prev_cuda, u_cuda);
        SWAP(v_prev_cuda, v_cuda);
        SWAP(dens_prev_cuda, dens_cuda);

        //// diffuse step
        float a = dt * visc * N * N;
        lin_solve << < 1, num_threads, 0, stream1 >> > (N, 1, u_cuda, u_prev_cuda, a, 1 + 4 * a, elementsPerThread); // diffuse u_cuda
        lin_solve << < 1, num_threads, 0, stream2 >> > (N, 2, v_cuda, v_prev_cuda, a, 1 + 4 * a, elementsPerThread); // diffuse v_cuda
        a = dt * diff * N * N;
        lin_solve << < 1, num_threads, 0, stream3 >> > (N, 0, dens_cuda, dens_prev_cuda, a, 1 + 4 * a, elementsPerThread); // diffuse dens_cuda
        hipDeviceSynchronize();


        // projection step (no swapping beforehand)
        project1 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, div_cuda, elementsPerThread);
        lin_solve << < 1, num_threads >> > (N, 0, p_cuda, div_cuda, 1, 4, elementsPerThread);
        project3 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, elementsPerThread);
        hipDeviceSynchronize();

        SWAP(u_prev_cuda, u_cuda);
        SWAP(v_prev_cuda, v_cuda);

        // advection step (velocity grid)
        advect << < 1, num_threads, 0, stream1 >> > (N, 1, u_cuda, u_prev_cuda, u_prev_cuda, v_prev_cuda, dt, elementsPerThread);
        advect << < 1, num_threads, 0, stream2 >> > (N, 2, v_cuda, v_prev_cuda, u_prev_cuda, v_prev_cuda, dt, elementsPerThread);
        hipDeviceSynchronize();


        // projection step (no swapping beforehand)
        project1 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, div_cuda, elementsPerThread);
        lin_solve << < 1, num_threads >> > (N, 0, p_cuda, div_cuda, 1, 4, elementsPerThread);
        project3 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, elementsPerThread);
        hipDeviceSynchronize();
    

        // advection step (density grid)
        SWAP(dens_prev_cuda, dens_cuda);
        advect << < 1, num_threads >> > (N, 0, dens_cuda, dens_prev_cuda, u_cuda, v_cuda, dt, elementsPerThread);
        hipDeviceSynchronize();

        // Display results if needed
        printf("Finished iteration %d:\n", i);
        if (display_output) {
            printFrameMatrices(dens_cuda, u_cuda, v_cuda, N);
            puts("\n");
        }
    }

    // Free Device space
    hipFree(u_cuda);
    hipFree(v_cuda);
    hipFree(u_prev_cuda);
    hipFree(v_prev_cuda);
    hipFree(dens_cuda);
    hipFree(dens_prev_cuda);
    hipFree(p_cuda);
    hipFree(div_cuda);
    

    // Destroy cuda streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);

    return 0;
}
