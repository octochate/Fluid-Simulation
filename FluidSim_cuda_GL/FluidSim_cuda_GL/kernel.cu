#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <string>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <time.h>
#include "freeglut.h"
#include <mutex>

/* macros */
#define IX(i,j) ((i)+(N+2)*(j))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}
#define MIN(x, y) (x > y) ? y : x
#define MAX(x, y) (x < y) ? y : x

static int allocate_data(void);

__global__ void add_source(float* x, float* s, float dt, int size)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x);

	if (idx < size) {
		x[idx] += dt * s[idx];
	}
}

__device__ void set_bnd(int N, int b, float* x, int index, int elementsPerThread)
{
	int i = index + 1;
	if (i > N + 1) {
		return;
	}

	int size = (N + 2) * (N + 2);

	while (i < (index + elementsPerThread) && i <= N + 1) {
		x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
		x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
		x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
		x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
		i++;
	}

	__syncthreads();

	if (index == 0) {
		x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
		x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
		x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
		x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
	}
}

__global__ void lin_solve(int N, int b, float* x, float* x0, float a, float c, int elementPerThread)
{
	int i, j, k, idxNew;
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

	if (idx >= N * N) {
		return;
	}

	for (k = 0; k < 20; k++)
	{
		idxNew = idx;
		while (idxNew < idx + elementPerThread && idxNew < N * N) {
			j = idxNew / N + 1;
			i = idxNew % N + 1;
			x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
			idxNew++;
		}
		__syncthreads();
		set_bnd(N, b, x, idx, elementPerThread);
	}
}

__global__ void advect(int N, int b, float* d, float* d0, float* u, float* v, float dt, int elementPerThread)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

	if (idx >= N * N) {
		return;
	}

	int idxNew = idx;
	while (idxNew < idx + elementPerThread && idxNew < N * N) {
		int j = idxNew / N + 1;
		int i = idxNew % N + 1;

		int i0, j0, i1, j1;
		float x, y, s0, t0, s1, t1, dt0;

		dt0 = dt * N;
		x = i - dt0 * u[IX(i, j)]; y = j - dt0 * v[IX(i, j)];
		if (x < 0.5f) x = 0.5f; if (x > N + 0.5f) x = N + 0.5f; i0 = (int)x; i1 = i0 + 1;
		if (y < 0.5f) y = 0.5f; if (y > N + 0.5f) y = N + 0.5f; j0 = (int)y; j1 = j0 + 1;
		s1 = x - i0; s0 = 1 - s1; t1 = y - j0; t0 = 1 - t1;

		d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
			s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);

		idxNew++;
	}

	__syncthreads();
	set_bnd(N, b, d, idx, elementPerThread);
}

__global__ void project1(int N, float* u, float* v, float* p, float* div, int elementPerThread)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

	if (idx >= N * N) {
		return;
	}

	int idxNew = idx;
	while (idxNew < idx + elementPerThread && idxNew < N * N) {
		int j = idxNew / N + 1;
		int i = idxNew % N + 1;

		div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / N;
		p[IX(i, j)] = 0;
		idxNew++;
	}

	__syncthreads();
	set_bnd(N, 0, div, idx, elementPerThread);
	set_bnd(N, 0, p, idx, elementPerThread);
}

__global__ void project3(int N, float* u, float* v, float* p, int elementPerThread)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) * elementPerThread;

	if (idx >= N * N) {
		return;
	}

	int idxNew = idx;
	while (idxNew < idx + elementPerThread && idxNew < N * N) {
		int j = idxNew / N + 1;
		int i = idxNew % N + 1;

		u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
		v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);

		idxNew++;
	}
	__syncthreads();
	set_bnd(N, 1, u, idx, elementPerThread);
	set_bnd(N, 2, v, idx, elementPerThread);
}


/* global variables */
hipStream_t stream1 = NULL, stream2 = NULL, stream3 = NULL;

volatile static int N, size;
volatile static float dt, diff, visc, force, source;
volatile static int display_velocity = 0, cuda_streams = 0, shoot_liquid = 0;

float numIntervals = 100;
float dtMax = 5;
float dtMin = 0.01;
float dt_del = (dtMax - dtMin) / numIntervals;
float viscMax = 1;
float viscMin = 0;
float visc_del = (viscMax - viscMin) / numIntervals;
float diffMax = 1.;
float diffMin = 0;
float diff_del = (diffMax - diffMin) / numIntervals;
float fluidAmountMax = 1000.;
float fluidAmountMin = 1.;
float fluidAmount_del = (fluidAmountMax - fluidAmountMin) / numIntervals;
float forceAmountMax = 5;
float forceAmountMin = 0.1;
float forceAmount_del = (forceAmountMax - forceAmountMin) / numIntervals;

// Parallelize computation :: TODO update values to match
int num_threads_source;
int num_blocks_source;
int num_threads;
int elementsPerThread;

// CPU variables
static float* u, * v, * dens;
static float* u_userInput, * v_userInput, * dens_userInput;

volatile int flag = 0;
std::mutex * guimutexPtr;

// GPU variables
float* u_cuda, * v_cuda, * u_prev_cuda, * v_prev_cuda;
float* p_cuda, * div_cuda;
float* dens_cuda, * dens_prev_cuda;

static int win_id;
static int win_x, win_y;
static int mouse_down[3];
static int omx, omy, mx, my;
static float xtext, ytext;


/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data(void)
{
	if (u_userInput) free(u_userInput);
	if (v_userInput) free(v_userInput);
	if (dens_userInput) free(dens_userInput);

	if (u) free(u);
	if (v) free(v);
	if (dens) free(dens);

	if (u_cuda) hipFree(u_cuda);
	if (v_cuda) hipFree(v_cuda);
	if (u_prev_cuda) hipFree(u_prev_cuda);
	if (v_prev_cuda) hipFree(v_prev_cuda);
	if (p_cuda) hipFree(p_cuda);
	if (div_cuda) hipFree(div_cuda);
	if (dens_cuda) hipFree(dens_cuda);
	if (dens_prev_cuda) hipFree(dens_prev_cuda);
}

static void clear_data(void)
{
	hipMemset(u_cuda, 0, sizeof(float) * size);
	hipMemset(v_cuda, 0, sizeof(float) * size);
	hipMemset(p_cuda, 0, sizeof(float) * size);
	hipMemset(div_cuda, 0, sizeof(float) * size);
	hipMemset(dens_cuda, 0, sizeof(float) * size);
	hipMemset(u_prev_cuda, 0, sizeof(float) * size);
	hipMemset(v_prev_cuda, 0, sizeof(float) * size);
	hipMemset(dens_prev_cuda, 0, sizeof(float) * size);

	for (int i = 0; i < size; i++) {
		u_userInput[i] = v_userInput[i] = dens_userInput[i] = 0.0f;
	}
}

static void destroy_streams(void) {
	// Destroy cuda streams
	if (stream1 == stream2 && stream2 == stream3) {
		hipStreamDestroy(stream1);
	}
	else {
		hipStreamDestroy(stream1);
		hipStreamDestroy(stream2);
		hipStreamDestroy(stream3);
	}

	stream1 = NULL;
	stream2 = NULL;
	stream3 = NULL;
}

static void create_streams(void) {
	if (stream1 != NULL) {
		return;
	}

	if (cuda_streams) {
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		hipStreamCreate(&stream3);
	}
	else {
		hipStreamCreate(&stream1);
		stream2 = stream1;
		stream3 = stream1;
	}
}

static void set_numThreads(int newValue) {
	if (newValue < 1) {
		newValue = 1;
	}
	if (newValue > 1024) {
		newValue = 1024;
	}

	num_threads = newValue;
	elementsPerThread = (N * N + 1) / num_threads;

}

static int set_gridSize(int newValue) {
	if (newValue < 32) {
		newValue = 32;
	}
	if (newValue > 1024) {
		newValue = 1024;
	}

	if (N == newValue) {
		return 1;
	}

	//hipDeviceReset();
	hipDeviceSynchronize();
	const std::lock_guard<std::mutex> lock(*guimutexPtr);
	free_data();
	N = newValue;
	size = (N + 2) * (N + 2);
	num_threads_source = (N + 2);
	num_blocks_source = (N + 2);
	set_numThreads(num_threads);
	return allocate_data();
}

static int allocate_data(void)
{
	// Allocate space for device copies
	u = (float*)malloc(size * sizeof(float));
	v = (float*)malloc(size * sizeof(float));
	dens = (float*)malloc(size * sizeof(float));

	u_userInput = (float*)malloc(size * sizeof(float));
	v_userInput = (float*)malloc(size * sizeof(float));
	dens_userInput = (float*)malloc(size * sizeof(float));

	// gpu copies
	hipMalloc(&u_cuda, sizeof(float) * size);
	hipMalloc(&v_cuda, sizeof(float) * size);
	hipMalloc(&p_cuda, sizeof(float) * size);
	hipMalloc(&div_cuda, sizeof(float) * size);
	hipMalloc(&u_prev_cuda, sizeof(float) * size);
	hipMalloc(&v_prev_cuda, sizeof(float) * size);
	hipMalloc(&dens_cuda, sizeof(float) * size);
	hipMalloc(&dens_prev_cuda, sizeof(float) * size);


	if (!u_userInput || !v_userInput || !dens_userInput || !dens || !u || !v || !u_cuda || !v_cuda || !u_prev_cuda || !v_prev_cuda || !dens_cuda || !dens_prev_cuda) {
		fprintf(stderr, "cannot allocate data\n");
		return (0);
	}

	return (1);
}


/*
  ----------------------------------------------------------------------
   OpenGL specific drawing routines
  ----------------------------------------------------------------------
*/

static void pre_display(void)
{
	glViewport(0, 0, win_x, win_y);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, 1.0, 0.0, 1.0);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT);
}

static void post_display(void)
{
	glutSwapBuffers();
}

static void draw_velocity(void)
{
	int i, j;
	float x, y, h;

	h = 1.0f / N;

	hipMemcpy(u, u_cuda, sizeof(float) * size, hipMemcpyDeviceToHost);
	hipMemcpy(v, v_cuda, sizeof(float) * size, hipMemcpyDeviceToHost);

	glColor3f(1.0f, 1.0f, 1.0f);
	glLineWidth(1.0f);

	glBegin(GL_LINES);

	for (i = 1; i <= N; i++) {
		x = (i - 0.5f) * h;
		for (j = 1; j <= N; j++) {
			y = (j - 0.5f) * h;

			glVertex2f(x, y);
			glVertex2f(x + u[IX(i, j)], y + v[IX(i, j)]);
		}
	}

	glEnd();
}

static void draw_density(void)
{
	int i, j;
	float x, y, h, d00, d01, d10, d11;

	h = 1.0f / N;
	hipMemcpy(dens, dens_cuda, sizeof(float) * size, hipMemcpyDeviceToHost);

	glBegin(GL_QUADS);

	for (i = 0; i <= N; i++) {
		x = (i - 0.5f) * h;
		for (j = 0; j <= N; j++) {
			y = (j - 0.5f) * h;

			d00 = dens[IX(i, j)];
			d01 = dens[IX(i, j + 1)];
			d10 = dens[IX(i + 1, j)];
			d11 = dens[IX(i + 1, j + 1)];

			glColor3f(d00, d00, d00 * 0); glVertex2f(x, y);
			glColor3f(d10, d10, d10 * 0); glVertex2f(x + h, y);
			glColor3f(d11, d11, d11 * 0); glVertex2f(x + h, y + h);
			glColor3f(d01, d01, d01 * 0); glVertex2f(x, y + h);
		}
	}

	glEnd();
}

/*
  ----------------------------------------------------------------------
   relates mouse movements to forces sources
  ----------------------------------------------------------------------
*/

static void get_from_UI(float* d, float* u, float* v)
{
	hipMemcpy(u_prev_cuda, u_userInput, sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(v_prev_cuda, v_userInput, sizeof(float) * size, hipMemcpyHostToDevice);;
	hipMemcpy(dens_prev_cuda, dens_userInput, sizeof(float) * size, hipMemcpyHostToDevice);
	return;
}

static void get_from_UI_CPU(void)
{
	int i, j;

	if (shoot_liquid) {
		i = (N + 2) / 5;
		j = (N + 2) / 5;

		u_userInput[IX(i, j)] = force;
		v_userInput[IX(i, j)] = force;
		dens_userInput[IX(i, j)] = source;
	}

	if (!mouse_down[0] && !mouse_down[2]) return;

	i = (int)((mx / (float)win_x) * N + 1);
	j = (int)(((win_y - my) / (float)win_y) * N + 1);

	if (i<1 || i>N || j<1 || j>N) return;

	if (mouse_down[0]) {
		u_userInput[IX(i, j)] += force * (mx - omx);
		v_userInput[IX(i, j)] += force * (omy - my);
	}

	if (mouse_down[2]) {
		dens_userInput[IX(i - 1, j)] = source;
		dens_userInput[IX(i + 1, j)] = source;
		dens_userInput[IX(i, j + 1)] = source;
		dens_userInput[IX(i, j - 1)] = source;
		dens_userInput[IX(i, j)] = source;
	}

	omx = mx;
	omy = my;

	return;
}

/*
  ----------------------------------------------------------------------
   GLUT callback routines
  ----------------------------------------------------------------------
*/
static void Sliders()
{
	const unsigned char a[50] = "Controls";
	const unsigned char b[50] = "Time Step    [a]";
	const unsigned char c[50] = "Viscosity    [s]";
	const unsigned char d[50] = "Diffussion   [d]";
	const unsigned char e[50] = "Fluid Amount [f]";
	const unsigned char f[50] = "Force Amount [g]";
	const unsigned char g[50] = "# threads: ";
	const unsigned char h[50] = "grid size: ";

	unsigned char numThreads[50];
	strcpy((char*) numThreads, std::to_string(num_threads).c_str());
	const unsigned char* numThreads_const = (const unsigned char*)numThreads;

	unsigned char gridSize[50];
	strcpy((char*)gridSize, std::to_string(N).c_str());
	const unsigned char* gridSize_const = (const unsigned char*)gridSize;

	const unsigned char cudaStreams[50] = "CUDA streams enabled";
	const unsigned char dispenser[50] = "Jet enabled";

	const unsigned char* aPtr = a;
	const unsigned char* bPtr = b;
	const unsigned char* cPtr = c;
	const unsigned char* dPtr = d;
	const unsigned char* ePtr = e;
	const unsigned char* fPtr = f;
	const unsigned char* gPtr = g;
	const unsigned char* hPtr = h;
	const unsigned char* streamsPtr = cudaStreams;
	const unsigned char* jetPtr = dispenser;

	// Drawing Sliders Text Fields
	glColor3f(1.0, 1.0, 1.0);
	glRasterPos2f(0.78125, 0.94921875);
	glutBitmapString(GLUT_BITMAP_9_BY_15, aPtr);
	glRasterPos2f(0.64453125, 0.90625000);
	glutBitmapString(GLUT_BITMAP_8_BY_13, bPtr);
	glRasterPos2f(0.64453125, 0.86718750);
	glutBitmapString(GLUT_BITMAP_8_BY_13, cPtr);
	glRasterPos2f(0.64453125, 0.82812500);
	glutBitmapString(GLUT_BITMAP_8_BY_13, dPtr);
	glRasterPos2f(0.64453125, 0.7890625);
	glutBitmapString(GLUT_BITMAP_8_BY_13, ePtr);
	glRasterPos2f(0.64453125, 0.7500000);
	glutBitmapString(GLUT_BITMAP_8_BY_13, fPtr);
	glRasterPos2f(0.78125 + 0.05, 0.7109375);
	glutBitmapString(GLUT_BITMAP_8_BY_13, gPtr);
	glRasterPos2f(0.78125 + 0.05 + 0.1, 0.7109375);
	glutBitmapString(GLUT_BITMAP_8_BY_13, numThreads_const);
	glRasterPos2f(0.78125 + 0.05, 0.6718750 + 0.025);
	glutBitmapString(GLUT_BITMAP_8_BY_13, hPtr);
	glRasterPos2f(0.78125 + 0.05 + 0.1, 0.6718750 + 0.025);
	glutBitmapString(GLUT_BITMAP_8_BY_13, gridSize_const);
	
	if (cuda_streams) {
		glRasterPos2f(0.78125, 0.6718750);
		glutBitmapString(GLUT_BITMAP_8_BY_13, streamsPtr);
	}
	if (shoot_liquid) {
		glRasterPos2f(0.78125, 0.6718750 - 0.025);
		glutBitmapString(GLUT_BITMAP_8_BY_13, jetPtr);
	}

	glRasterPos2f(0., 0.);

	glBegin(GL_LINES);
	glColor3f(1.0, 1.0, 1.0);

	// Draw slider boxes.
	for (int i = 0; i < 5; i++)
	{
		// Compute heights.
		float heightTop = 1. - (38. + (float)i * 20.) / 512.;
		float heightBottom = 1. - (49. + (float)i * 20.) / 512.;
		glVertex2d(0.83984375, heightTop);
		glVertex2d(0.99609375, heightTop);
		glVertex2d(0.83984375, heightBottom);
		glVertex2d(0.99609375, heightBottom);
		glVertex2d(0.83984375, heightTop);
		glVertex2d(0.83984375, heightBottom);
		glVertex2d(0.99609375, heightTop);
		glVertex2d(0.99609375, heightBottom);
	}


	// Fill In Sliders
	float sliderStart = 0.83984375;
	float sliderEnd = 0.99609375;
	// Variable bounds.

	// Compute dynamic slider fill.
	float dtSliderEnd = ((dt / dtMax) * 0.15625) + sliderStart;
	float viscSliderEnd = ((visc / viscMax) * 0.15625) + sliderStart;
	float diffSliderEnd = ((diff / diffMax) * 0.15625) + sliderStart;
	float fluidAmountSliderEnd = ((source / fluidAmountMax) * 0.15625) + sliderStart;
	float forceAmountSliderEnd = ((force / forceAmountMax) * 0.15625) + sliderStart;

	for (float i = sliderStart; i <= sliderEnd; i += 0.001)
	{
		float heightTop = 0.0;
		float heightBottom = 0.0;
		if (i <= dtSliderEnd)
		{
			heightTop = 1. - (38. + 0. * 20.) / 512.;
			heightBottom = 1. - (49. + 0. * 20.) / 512.;
			glVertex2d(i, heightTop);
			glVertex2d(i, heightBottom);
		}
		if (i <= viscSliderEnd)
		{
			heightTop = 1. - (38. + 1. * 20.) / 512.;
			heightBottom = 1. - (49. + 1. * 20.) / 512.;
			glVertex2d(i, heightTop);
			glVertex2d(i, heightBottom);
		}
		if (i <= diffSliderEnd)
		{
			heightTop = 1. - (38. + 2. * 20.) / 512.;
			heightBottom = 1. - (49. + 2. * 20.) / 512.;
			glVertex2d(i, heightTop);
			glVertex2d(i, heightBottom);
		}
		if (i <= fluidAmountSliderEnd)
		{
			heightTop = 1. - (38. + 3. * 20.) / 512.;
			heightBottom = 1. - (49. + 3. * 20.) / 512.;
			glVertex2d(i, heightTop);
			glVertex2d(i, heightBottom);
		}
		if (i <= forceAmountSliderEnd)
		{
			heightTop = 1. - (38. + 4. * 20.) / 512.;
			heightBottom = 1. - (49. + 4. * 20.) / 512.;
			glVertex2d(i, heightTop);
			glVertex2d(i, heightBottom);
		}
	}
	glEnd();
}

static void key_func(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 'c':
	case 'C':
		clear_data();
		break;

	case 'v':
	case 'V':
		display_velocity = !display_velocity;
		break;

	case 'a':
		dt = MIN(dtMax, dt+ dt_del);
		printf("dt is now %f\n", dt);
		break;

	case 'A':
		dt = MAX(dtMin, dt - dt_del);
		printf("dt is now %f\n", dt);
		break;
	case 's':
		visc = MIN(viscMax, visc + visc_del);
		printf("visc is now %f\n", visc);
		break;

	case 'S':
		visc = MAX(viscMin, visc - visc_del);
		printf("visc is now %f\n", visc);
		break;
	case 'd':
		diff = MIN(diffMax, diff + diff_del);
		printf("diff is now %f\n", diff);
		break;

	case 'D':
		diff = MAX(diffMin, diff - diff_del);
		printf("diff is now %f\n", diff);
		break;
	case 'f':
		source = MIN(fluidAmountMax, source + fluidAmount_del);
		printf("fluidAmount is now %f\n", source);
		break;

	case 'F':
		source = MAX(fluidAmountMin, source - fluidAmount_del);
		printf("fluidAmount is now %f\n", source);
		break;
	case 'g':
		force = MIN(forceAmountMax, force + forceAmount_del);
		printf("forceAmount is now %f\n", force);
		break;

	case 'G':
		force = MAX(forceAmountMin, force - forceAmount_del);
		printf("forceAmount is now %f\n", force);
		break;

	case 'e':
	case 'E':
		destroy_streams();
		cuda_streams = !cuda_streams;
		create_streams();
		break;
	case 'w':
	case 'W':
		shoot_liquid = !shoot_liquid;
		break;
	case '1':
		set_numThreads(num_threads * 2);
		break;
	case '2':
		set_numThreads(num_threads / 2);
		break;
	case '3':
		if (!set_gridSize(N + 100)) exit(1);
		clear_data();
		break;
	case '4':
		if (!set_gridSize(N - 100)) exit(1);
		clear_data();
		break;
	}

}

static void mouse_func(int button, int state, int x, int y)
{
	omx = mx = x;
	omx = my = y;

	mouse_down[button] = state == GLUT_DOWN;
}

static void motion_func(int x, int y)
{
	mx = x;
	my = y;
}

static void reshape_func(int width, int height)
{
	glutSetWindow(win_id);
	glutReshapeWindow(width, height);

	win_x = width;
	win_y = height;
}

static void idle_func(void)
{
	// have copy of host data
	get_from_UI(dens_prev_cuda, u_prev_cuda, v_prev_cuda);
	hipDeviceSynchronize();

	////// Velocity timestep parallelization
	add_source << < num_blocks_source, num_threads_source >> > (u_cuda, u_prev_cuda, dt, size);
	add_source << < num_blocks_source, num_threads_source >> > (v_cuda, v_prev_cuda, dt, size);
	add_source << < num_blocks_source, num_threads_source >> > (dens_cuda, dens_prev_cuda, dt, size);

	for (int i = 0; i < size; i++) {
		u_userInput[i] = v_userInput[i] = dens_userInput[i] = 0.0f;
	}

	SWAP(u_prev_cuda, u_cuda);
	SWAP(v_prev_cuda, v_cuda);
	SWAP(dens_prev_cuda, dens_cuda);
	hipDeviceSynchronize();

	

	//// diffuse
	float a = dt * visc * N * N;
	lin_solve << < 1, num_threads, 0, stream1 >> > (N, 1, u_cuda, u_prev_cuda, a, 1 + 4 * a, elementsPerThread); // diffuse u_cuda
	lin_solve << < 1, num_threads, 0, stream2 >> > (N, 2, v_cuda, v_prev_cuda, a, 1 + 4 * a, elementsPerThread); // diffuse v_cuda

	a = dt * diff * N * N;
	lin_solve << < 1, num_threads, 0, stream3 >> > (N, 0, dens_cuda, dens_prev_cuda, a, 1 + 4 * a, elementsPerThread); // diffuse dens_cuda
	
	hipDeviceSynchronize();

	// projection step (no swapping beforehand)
	project1 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, div_cuda, elementsPerThread);
	lin_solve << < 1, num_threads >> > (N, 0, p_cuda, div_cuda, 1, 4, elementsPerThread);
	project3 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, elementsPerThread);
	SWAP(u_prev_cuda, u_cuda);
	SWAP(v_prev_cuda, v_cuda);
	hipDeviceSynchronize();

	advect << < 1, num_threads, 0, stream1 >> > (N, 1, u_cuda, u_prev_cuda, u_prev_cuda, v_prev_cuda, dt, elementsPerThread);
	advect << < 1, num_threads, 0, stream2 >> > (N, 2, v_cuda, v_prev_cuda, u_prev_cuda, v_prev_cuda, dt, elementsPerThread);
	hipDeviceSynchronize();

	// projection step (no swapping beforehand)
	project1 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, div_cuda, elementsPerThread);
	lin_solve << < 1, num_threads >> > (N, 0, p_cuda, div_cuda, 1, 4, elementsPerThread);
	project3 << < 1, num_threads >> > (N, u_cuda, v_cuda, p_cuda, elementsPerThread);
	SWAP(dens_prev_cuda, dens_cuda);
	hipDeviceSynchronize();

	// Density timestep parallelization
	advect << < 1, num_threads >> > (N, 0, dens_cuda, dens_prev_cuda, u_cuda, v_cuda, dt, elementsPerThread);

	glutSetWindow(win_id);
	glutPostRedisplay();
}

static void display_func(void)
{
	pre_display();

	if (display_velocity) draw_velocity();
	else		draw_density();

	Sliders();
	post_display();
}


/*
  ----------------------------------------------------------------------
   open_glut_window --- open a glut compatible window and set callbacks
  ----------------------------------------------------------------------
*/

static void open_glut_window(void)
{
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);

	glutInitWindowPosition(0, 0);
	glutInitWindowSize(win_x, win_y);
	win_id = glutCreateWindow("Fluid Simulation");

	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT);
	glutSwapBuffers();
	glClear(GL_COLOR_BUFFER_BIT);
	glutSwapBuffers();

	pre_display();

	glutKeyboardFunc(key_func);
	glutMouseFunc(mouse_func);
	glutMotionFunc(motion_func);
	// glutReshapeFunc ( reshape_func );
	glutIdleFunc(idle_func);
	glutDisplayFunc(display_func);
}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main(int argc, char** argv)
{
	glutInit(&argc, argv);

	if (argc != 1 && argc != 6) {
		fprintf(stderr, "usage : %s N dt diff visc force source\n", argv[0]);
		fprintf(stderr, "where:\n"); \
			fprintf(stderr, "\t N      : grid resolution\n");
		fprintf(stderr, "\t dt     : time step\n");
		fprintf(stderr, "\t diff   : diffusion rate of the density\n");
		fprintf(stderr, "\t visc   : viscosity of the fluid\n");
		fprintf(stderr, "\t force  : scales the mouse movement that generate a force\n");
		fprintf(stderr, "\t source : amount of density that will be deposited\n");
		exit(1);
	}

	if (argc == 1) {
		//N = 128;
		dt = 1.0f;
		diff = 0.0f;
		visc = 0.0f;
		force = 1.0f;
		source = 100.0f;
		fprintf(stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
			N, dt, diff, visc, force, source);
	}
	else {
		//N = atoi(argv[1]);
		dt = atof(argv[2]);
		diff = atof(argv[3]);
		visc = atof(argv[4]);
		force = atof(argv[5]);
		source = atof(argv[6]);
	}

	printf("\n\nHow to use this demo:\n\n");
	printf("\t Add densities with the right mouse button\n");
	printf("\t Add velocities with the left mouse button and dragging the mouse\n");
	printf("\t Toggle density/velocity display with the 'v' key\n");
	printf("\t Clear the simulation by pressing the 'c' key\n");
	printf("\t Quit by pressing the 'q' key\n");

	set_numThreads(1024);
	display_velocity = 0;

	std::mutex guimutex;
	guimutexPtr = &guimutex;

	if (!set_gridSize(300)) exit(1);
	clear_data();
	create_streams();

	
	// A mutex ensures orderly access to std::cout from multiple threads.
	std::thread t1([&guimutex]() {
		while (!flag) {
			{
				const std::lock_guard<std::mutex> lock(guimutex);
				get_from_UI_CPU();
			}
			std::this_thread::sleep_for(std::chrono::milliseconds(25));
		}
		});


	win_x = 800;
	win_y = 800;
	open_glut_window();

	glutMainLoop();

	//stop GUI thread
	flag = 1;
	t1.join();

	// Free Device space
	hipFree(u_cuda);
	hipFree(v_cuda);
	hipFree(u_prev_cuda);
	hipFree(dens_cuda);
	hipFree(dens_prev_cuda);
	hipFree(v_prev_cuda);

	destroy_streams();

	exit(0);
}