#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdio.h>
#include "glew.h"
#include "cuda_gl_interop.h"
#include "freeglut.h"


static int windowID;
static float width, height;
// static int widthINT, heightINT;
__device__ float *arrayCUDA;
float *arrayHOST;
int *parrayCUDA;
float *parrayHOST;
static int arraySize, arrayDimension;

static float dt, visc;

static void Sliders()
{
    // Drawing Sliders Text Fields
    glColor3f(0.0, 0.0, 0.0);
    glRasterPos2f(408., 26.);
    glutBitmapString(GLUT_BITMAP_9_BY_15, "Controls");
    glRasterPos2f(350., 48.);
    glutBitmapString(GLUT_BITMAP_8_BY_13, "Time Step");
    glRasterPos2f(350., 68.);
    glutBitmapString(GLUT_BITMAP_8_BY_13, "Viscosity");
    glRasterPos2f(350., 88.);
    glutBitmapString(GLUT_BITMAP_8_BY_13, "Vorticity");
    glRasterPos2f(0., 0.);

    glBegin(GL_LINES);

    // Time Step
    glVertex2d(430., 38.);
    glVertex2d(510., 38.);
    glVertex2d(430., 49.);
    glVertex2d(510., 49.);
    glVertex2d(430., 38.);
    glVertex2d(430., 49.);
    glVertex2d(510., 38.);
    glVertex2d(510., 49.);

    // Viscosity
    glVertex2d(430., 58.);
    glVertex2d(510., 58.);
    glVertex2d(430., 69.);
    glVertex2d(510., 69.);
    glVertex2d(430., 58.);
    glVertex2d(430., 69.);
    glVertex2d(510., 58.);
    glVertex2d(510., 69.);

    // Vorticity
    glVertex2d(430., 78.);
    glVertex2d(510., 78.);
    glVertex2d(430., 89.);
    glVertex2d(510., 89.);
    glVertex2d(430., 78.);
    glVertex2d(430., 89.);
    glVertex2d(510., 78.);
    glVertex2d(510., 89.);

    // Fill In Sliders
    float sliderStart = 430.;
    float sliderEnd = 510.;
    if (dt > 30.)
        dt = 30.;
    if (visc > 1.)
        visc = 1.;
    float dtSliderEnd = ((dt / 30.) * 80.) + sliderStart;
    float viscSliderEnd = ((visc / 1.) * 80.) + sliderStart;
    for (float i = sliderStart; i <= sliderEnd; i++)
    {
        if (i <= dtSliderEnd)
        {
            glVertex2d(i, 38.);
            glVertex2d(i, 49.);
        }
        if (i <= viscSliderEnd)
        {
            glVertex2d(i, 58.);
            glVertex2d(i, 69.);
        }
    }
    glEnd();
}

__global__ static void color_Array()
{
    // int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < 512; i++)
    {
        for (int j = 0; j < 512; j++)
        {
            arrayCUDA[i + j * 512] = 0.5;
        }
    }
}
static void draw_Array()
{
    glBegin(GL_POINTS);

    /////// This is where the problem lies
    // How to copy a __device__ array to the __host__ array.
    hipMemcpyFromSymbol(arrayHOST, HIP_SYMBOL((const char *))"arrayCUDA", arraySize, 0, hipMemcpyDeviceToHost);
    ///////
    for (int i = 0; i < 512; i++)
    {
        for (int j = 0; j < 512; j++)
        {
            float color = arrayHOST[i + j * 512];
            glColor3f(color, color, color);
            glVertex2i(i, j);
        }
    }
    glEnd();
}

static void pre_display(void)
{
    glViewport(0, 0, 512, 512);
    glLoadIdentity();
    gluOrtho2D(0.0, 512., 512., 0.0);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
}

static void post_display(void)
{
    glutSwapBuffers();
}


static void key_func(unsigned char key, int x, int y)
{
    switch (key)
    {
    case 'w':
    case 'W':
        dt += 0.1;
        printf("dt is now %f\n", dt);
        break;

    case 's':
    case 'S':
        dt -= 0.1;
        printf("dt is now %f\n", dt);
        break;

    case 'e':
    case 'E':
        visc += 0.05;
        printf("visc is now %f\n", visc);
        break;

    case 'd':
    case 'D':
        visc -= 0.05;
        printf("visc is now %f\n", visc);
        break;
    }
}

static void idle_func(void)
{
    glutSetWindow(windowID);
    glutPostRedisplay();
}

static void display_func(void)
{
    pre_display();
    color_Array<<<1,1>>>();
    hipDeviceSynchronize();
    draw_Array();
    Sliders();
    post_display();
}

static void open_glut_window(void)
{
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(512, 512);
    windowID = glutCreateWindow("Fluid Simulation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
    glutSwapBuffers();
    glClear(GL_COLOR_BUFFER_BIT);
    glutSwapBuffers();
    pre_display();
    glutKeyboardFunc(key_func);
    glutIdleFunc(idle_func);
    glutDisplayFunc(display_func);
}

int main(int argc, char **argv)
{
    glutInit(&argc, argv);
    width = 512.;
    height = 512.;
    dt = 0.1;
    visc = 0.;
    arrayDimension = width * height;
    arraySize = arrayDimension * sizeof(float);
    hipMalloc(&arrayCUDA, arraySize);
    arrayHOST = (float *)malloc(arraySize);
    std::cout << "arrayCUDA " << &arrayCUDA << "\n";
    std::cout << "arrayCUDAP " << parrayCUDA << "\n";
    std::cout << "arrayCUDAPP " << &parrayCUDA << "\n";
    open_glut_window();
    glutMainLoop();
    // free(arrayCUDA);
    free(arrayHOST);
    exit(0);
}
