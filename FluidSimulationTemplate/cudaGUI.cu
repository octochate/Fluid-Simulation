#include "hip/hip_runtime.h"
// #include <stdlib.h>
// #include <stdio.h>
// #include "glut.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdio.h>
#include "glew.h"
#include "cuda_gl_interop.h"
#include "freeglut.h"


static int windowID;
static float width, height;
// static int widthINT, heightINT;
__device__ float *arrayCUDA;
float *arrayHOST;
int *parrayCUDA;
float *parrayHOST;
static int arraySize, arrayDimension;

static float dt, visc;

static void RenderString()
{
    glColor3f(1.0, 1.0, 1.0);
    // glColor3f(0.0, 0.0, 0.0);
    glRasterPos2f(408., 26.);
    glutBitmapString(GLUT_BITMAP_9_BY_15, (const unsigned char *)"Controls");
    glRasterPos2f(350., 48.);
    glutBitmapString(GLUT_BITMAP_8_BY_13, (const unsigned char *)"Time Step");
    glRasterPos2f(350., 68.);
    glutBitmapString(GLUT_BITMAP_8_BY_13, (const unsigned char *)"Viscosity");
    glRasterPos2f(350., 88.);
    glutBitmapString(GLUT_BITMAP_8_BY_13, (const unsigned char *)"Vorticity");
    glRasterPos2f(0., 0.);

    glBegin(GL_LINES);
    glVertex2d(430., 38.);
    glVertex2d(510., 38.);
    glVertex2d(430., 49.);
    glVertex2d(510., 49.);
    glVertex2d(430., 38.);
    glVertex2d(430., 49.);
    glVertex2d(510., 38.);
    glVertex2d(510., 49.);
    float sliderStart = 430.;
    float sliderEnd = 510.;
    if (dt > 30.)
        dt = 30.;
    if (visc > 1.)
        visc = 1.;
    float dtSliderEnd = ((dt / 30.) * 80.) + sliderStart;
    float viscSliderEnd = ((visc / 1.) * 80.) + sliderStart;
    for (float i = sliderStart; i <= sliderEnd; i++)
    {
        if (i <= dtSliderEnd)
        {
            glVertex2d(i, 38.);
            glVertex2d(i, 49.);
        }
        if (i <= viscSliderEnd)
        {
            glVertex2d(i, 58.);
            glVertex2d(i, 69.);
        }
    }

    // First
    glVertex2d(430., 58.);
    glVertex2d(510., 58.);
    glVertex2d(430., 69.);
    glVertex2d(510., 69.);
    glVertex2d(430., 58.);
    glVertex2d(430., 69.);
    glVertex2d(510., 58.);
    glVertex2d(510., 69.);
    // First
    glVertex2d(430., 78.);
    glVertex2d(510., 78.);
    glVertex2d(430., 89.);
    glVertex2d(510., 89.);
    glVertex2d(430., 78.);
    glVertex2d(430., 89.);
    glVertex2d(510., 78.);
    glVertex2d(510., 89.);

    glEnd();
}

__global__ static void color_Array()
{
    // arrayCUDA[65536] = 0.5;
    // int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < 512; i++)
    {
        for (int j = 0; j < 512; j++)
        {
            arrayCUDA[i + j * 512] = 0.5;
        }
    }
}
static void draw_Array()
{
    glBegin(GL_POINTS);
    // hipMemcpy(arrayHOST, parrayCUDA, arraySize, hipMemcpyDeviceToHost);
    // hipMemcpyFromSymbol(arrayHOST, HIP_SYMBOL("arrayCUDA"), arraySize, 0, hipMemcpyDeviceToHost);
    // hipMemcpyFromSymbol((void**)&arrayHOST, "arrayCUDA", arraySize, 0, hipMemcpyDeviceToHost);
    // parrayCUDA = NULL;
    // hipMemcpyFromSymbol((void**)&parrayCUDA, "arrayCUDA", arraySize, 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(arrayHOST, HIP_SYMBOL((const char *))"arrayCUDA", arraySize, 0, hipMemcpyDeviceToHost);
    // hipMemcpy(arrayHOST, parrayCUDA, arraySize, hipMemcpyDeviceToHost);
    // printf("HOST %f\n", arrayHOST[0]);

    for (int i = 0; i < 512; i++)
    {
        for (int j = 0; j < 512; j++)
        {
            float color = arrayHOST[i + j * 512];
            glColor3f(color, color, color);
            glVertex2i(i, j);
        }
    }
    glEnd();
}

static void pre_display(void)
{
    glViewport(0, 0, 512, 512);
    glLoadIdentity();
    gluOrtho2D(0.0, 512., 512., 0.0);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
}

static void post_display(void)
{
    glutSwapBuffers();
}


static void key_func(unsigned char key, int x, int y)
{
    switch (key)
    {
    case 'w':
    case 'W':
        dt += 0.1;
        printf("dt is now %f\n", dt);
        break;

    case 's':
    case 'S':
        dt -= 0.1;
        printf("dt is now %f\n", dt);
        break;

    case 'e':
    case 'E':
        visc += 0.05;
        printf("visc is now %f\n", visc);
        break;

    case 'd':
    case 'D':
        visc -= 0.05;
        printf("visc is now %f\n", visc);
        break;
    }
}

static void idle_func(void)
{
    glutSetWindow(windowID);
    glutPostRedisplay();
}

static void display_func(void)
{
    pre_display();
    color_Array<<<1,1>>>();
    hipDeviceSynchronize();
    draw_Array();
    RenderString();
    post_display();
}

static void open_glut_window(void)
{
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(512, 512);
    windowID = glutCreateWindow("Fluid Simulation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
    glutSwapBuffers();
    glClear(GL_COLOR_BUFFER_BIT);
    glutSwapBuffers();
    pre_display();
    glutKeyboardFunc(key_func);
    glutIdleFunc(idle_func);
    glutDisplayFunc(display_func);
}

int main(int argc, char **argv)
{
    glutInit(&argc, argv);
    width = 512.;
    height = 512.;
    dt = 0.1;
    visc = 0.;
    arrayDimension = width * height;
    arraySize = arrayDimension * sizeof(float);
    hipMalloc(&arrayCUDA, arraySize);
    arrayHOST = (float *)malloc(arraySize);
    // std::cout << "arrayCUDAPPP " << *parrayCUDA << "\n";
    // arrayHOST = NULL;
    // parrayCUDA = &arrayCUDA[0];
    std::cout << "arrayCUDA " << &arrayCUDA << "\n";
    std::cout << "arrayCUDAP " << parrayCUDA << "\n";
    std::cout << "arrayCUDAPP " << &parrayCUDA << "\n";
    open_glut_window();
    glutMainLoop();
    // free(arrayCUDA);
    free(arrayHOST);
    exit(0);
}