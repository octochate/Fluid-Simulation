#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

#define IX(i,j) ((i)+(N+2)*(j))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}
#define FOR_EACH_CELL for ( i=1 ; i<=N ; i++ ) { for ( j=1 ; j<=N ; j++ ) {
#define END_FOR }}

__device__ void add_source(int N, float* x, float* s, float dt, int size)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    if (idx < size) {
        x[idx] += dt * s[idx];
    }
}

__device__ void set_bnd(int N, int b, float* x)
{
    int i;

    for (i = 1; i <= N; i++)
    {
        x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
        x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
        x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
        x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
    }
    x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
    x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
    x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
}
__device__ void lin_solve(int N, int b, float* x, float* x0, float a, float c)
{
    int i, j, k;
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    j = idx / N;
    i = idx % N;

    for (k = 0; k < 20; k++)
    {
         x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
         __syncthreads();
         if (idx == 0) {
             set_bnd(N, b, x);
         }
    }
}

__global__ void diffuse(int N, int b, float* x, float* x0, float diff, float dt)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    float a = dt * diff * N * N;
    lin_solve(N, b, x, x0, a, 1 + 4 * a);
}

__global__ void advect(int N, int b, float* d, float* d0, float* u, float* v, float dt)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    int i, j, i0, j0, i1, j1;
    float x, y, s0, t0, s1, t1, dt0;

    dt0 = dt * N;
    FOR_EACH_CELL
        x = i - dt0 * u[IX(i, j)]; y = j - dt0 * v[IX(i, j)];
    if (x < 0.5f) x = 0.5f; if (x > N + 0.5f) x = N + 0.5f; i0 = (int)x; i1 = i0 + 1;
    if (y < 0.5f) y = 0.5f; if (y > N + 0.5f) y = N + 0.5f; j0 = (int)y; j1 = j0 + 1;
    s1 = x - i0; s0 = 1 - s1; t1 = y - j0; t0 = 1 - t1;
    d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
        s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
    END_FOR
        set_bnd(N, b, d);
}


__global__ void project(int N, float* u, float* v, float* p, float* div)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);

    int i, j;

    FOR_EACH_CELL
        div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] + v[IX(i, j + 1)] - v[IX(i, j - 1)]) / N;
    p[IX(i, j)] = 0;
    END_FOR
        set_bnd(N, 0, div); set_bnd(N, 0, p);

    lin_solve(N, 0, p, div, 1, 4);

    FOR_EACH_CELL
        u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
    END_FOR
        set_bnd(N, 1, u); set_bnd(N, 2, v);
}

void get_from_UI(float* d, float* u, float* v, int force, int source, int N)
{
    int i, j, size = (N + 2) * (N + 2);

    for (i = 0; i < size; i++)
    {
        u[i] = v[i] = d[i] = 0.0f;
    }

    i = N / 2;
    j = N / 2;

    if (i < 1 || i > N || j < 1 || j > N)
        return;

    u[IX(i, j)] = force * (1);
    v[IX(i, j)] = force * (1);
    d[IX(i, j)] = source;

    return;
}

int main(int argc, char* argv[]) {
       
    int N = 4;
    float dt = 0.1f, diff = 0.0f, visc = 0.0f;
    float force = 0.5f, source = 100.0f;

    // Host copies of data
    float* u, * v, * u_prev, * v_prev;
    float* dens, * dens_prev;

    // Allocate space for host
    int size = (N + 2) * (N + 2);
    u = (float*)malloc(size * sizeof(float));
    v = (float*)malloc(size * sizeof(float));
    u_prev = (float*)malloc(size * sizeof(float));
    v_prev = (float*)malloc(size * sizeof(float));
    dens = (float*)malloc(size * sizeof(float));
    dens_prev = (float*)malloc(size * sizeof(float));

    // Device copies of data
    float* u_cuda, * v_cuda, * u_prev_cuda, * v_prev_cuda;
    float* dens_cuda, * dens_prev_cuda;
    
    // Allocate space for device copies
    hipMalloc((void**)&u_cuda, sizeof(float));
    hipMalloc((void**)&v_cuda, sizeof(float));
    hipMalloc((void**)&u_prev_cuda, sizeof(float));
    hipMalloc((void**)&v_prev_cuda, sizeof(float));
    hipMalloc((void**)&dens_cuda, sizeof(float));
    hipMalloc((void**)&dens_prev_cuda, sizeof(float));

    // have copy of host data
    get_from_UI(dens, u_cuda, v_cuda, force, source, N);

    // Copy data from host to device
    hipMemcpy(u_cuda, u, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v_cuda, u, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(u_prev_cuda, u, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v_prev_cuda, u, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dens, u, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dens_prev_cuda, u, size * sizeof(float), hipMemcpyHostToDevice);

    // Parallelize computation :: TODO update values to match
    int num_threads = 1024;
    int num_blocks = 1;
    
    // Velocity timestep parallelization
    add_source(N, u_cuda, u_prev_cuda, dt, size);
    add_source(N, v_cuda, v_prev_cuda, dt, size);
    add_source(N, dens_cuda, dens_prev_cuda, dt, size);
    hipDeviceSynchronize();

    SWAP(u_prev_cuda, u_cuda);
    SWAP(v_prev_cuda, v_cuda);
    SWAP(dens_prev_cuda, dens_cuda);
    hipDeviceSynchronize();

    diffuse(N, 1, u_cuda, u_prev_cuda, visc, dt);
    diffuse(N, 2, v_cuda, v_prev_cuda, visc, dt);
    diffuse(N, 0, dens_cuda, dens_prev_cuda, diff, dt);
    hipDeviceSynchronize();

    project(N, u_cuda, v_cuda, u_prev_cuda, v_prev_cuda);
    SWAP(dens_prev_cuda, dens_cuda);
    hipDeviceSynchronize();

    SWAP(u_prev_cuda, u_cuda);
    SWAP(v_prev_cuda, v_cuda);
    hipDeviceSynchronize();

    advect(N, 1, u_cuda, u_prev_cuda, u_prev_cuda, v_prev_cuda, dt);
    advect(N, 2, v_cuda, v_prev_cuda, u_prev_cuda, v_prev_cuda, dt);
    hipDeviceSynchronize();

    project(N, u_cuda, v_cuda, u_prev_cuda, v_prev_cuda);
    hipDeviceSynchronize();

    // Density timestep parallelization
    advect(N, 0, dens_cuda, dens_prev_cuda, u_cuda, v_cuda, dt);
    hipDeviceSynchronize();

    // Copy result back to host

    // Free Device space
    hipFree(u_cuda);
    hipFree(v_cuda);
    hipFree(u_prev_cuda);
    hipFree(dens_cuda);
    hipFree(dens_prev_cuda);
    hipFree(v_prev_cuda);

    // Free Host space
    free(u); 
    free(v); 
    free(u_prev);
    free(v_prev); 
    free(dens); 
    free(dens_prev);

	return 0;
}
